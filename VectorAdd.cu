#include "hip/hip_runtime.h"
#include <iostream>
#include <Windows.h>
#include <hip/hip_runtime.h>

using namespace std;

// Keyword: __global__
// Function: A[]+B[] --> C[]
// Where, numElements is the array size of A, B and C
__global__ void vectorAdd(float* A, float* B, float* C, int numElements)
{
	// Each thread(i) does its own work
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		C[i] = A[i] + B[i];
	}
}

int main(void)
{
	DWORD start = GetTickCount();

	int numElements = 50000;
	size_t size = numElements * sizeof(float);

	cout << "Vector add: A+B-->C, where vector size=" << numElements << endl;

	// Maclloc on the host and randomly initialization
	float *h_A = new float[numElements];
	float *h_B = new float[numElements];
	float *h_C = new float[numElements];
	for (int i = 0; i < numElements; ++i)
	{
		h_A[i] = rand() / (float)RAND_MAX;
		h_B[i] = rand() / (float)RAND_MAX;
	}


	//hipError_t err = hipSuccess;
	//Malloc on the CUDA device
	float *d_A = NULL;
	hipMalloc((void **)&d_A, size);
	float *d_B = NULL;
	hipMalloc((void **)&d_B, size);
	float *d_C = NULL;
	hipMalloc((void **)&d_C, size);

	// COPY A&B: HOST--->CUDA
	cout << "Copy data:  [host memory] ---> [CUDA device]" << endl;
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	// CUDA Compute
	int threadsPerBlock = 256;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;  //�� ��һ����
	cout << "CUDA kernel launch with [" << blocksPerGrid << "(blocks) * "<< threadsPerBlock << "(threads)]." << endl;
	vectorAdd <<<blocksPerGrid, threadsPerBlock >>>(d_A, d_B, d_C, numElements);

	// COPY C: HOST<--CUDA
	cout << "Copy data:  [host memory] <--- [CUDA device] " << endl;
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	// Verify A+B==C
	cout << "Verifying the results [A+B==C] ..." << endl;
	for (int i = 0; i < numElements; ++i)
	{
		if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
		{
			cerr << "Result verification failed at element " << i << "!" << endl;
			exit(EXIT_FAILURE);
		}
	}

	cout << "Test PASSED" << endl;

	// Free memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	delete[] h_A;
	delete[] h_B;
	delete[] h_C;

	hipDeviceReset();

	DWORD finish = GetTickCount();

	cout << "Time taken:" << finish - start << endl;

	system("pause");
	return 0;
}